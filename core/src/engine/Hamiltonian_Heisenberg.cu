
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#define _USE_MATH_DEFINES
#include <cmath>

#include <Eigen/Dense>

#include <engine/Hamiltonian_Heisenberg.hpp>
#include <engine/Vectormath.hpp>
#include <data/Spin_System.hpp>
#include <utility/Constants.hpp>

using std::vector;
using std::function;

using namespace Data;
using namespace Utility;

namespace Engine
{
	Hamiltonian_Heisenberg::Hamiltonian_Heisenberg(
			scalarfield mu_s,
			intfield external_field_index, scalarfield external_field_magnitude, vectorfield external_field_normal,
			intfield anisotropy_index, scalarfield anisotropy_magnitude, vectorfield anisotropy_normal,
			std::vector<indexPairs> Exchange_indices, std::vector<scalarfield> Exchange_magnitude,
			std::vector<indexPairs> DMI_indices, std::vector<scalarfield> DMI_magnitude, std::vector<vectorfield> DMI_normal,
			std::vector<indexPairs> DD_indices, std::vector<scalarfield> DD_magnitude, std::vector<vectorfield> DD_normal,
			std::vector<indexQuadruplets> quadruplet_indices, std::vector<scalarfield> quadruplet_magnitude,
			std::vector<bool> boundary_conditions
	) :
		Hamiltonian(boundary_conditions),
		mu_s(mu_s),
		external_field_index(external_field_index), external_field_magnitude(external_field_magnitude), external_field_normal(external_field_normal),
		anisotropy_index(anisotropy_index), anisotropy_magnitude(anisotropy_magnitude), anisotropy_normal(anisotropy_normal),
		Exchange_indices(Exchange_indices), Exchange_magnitude(Exchange_magnitude),
		DMI_indices(DMI_indices), DMI_magnitude(DMI_magnitude), DMI_normal(DMI_normal),
		DD_indices(DD_indices), DD_magnitude(DD_magnitude), DD_normal(DD_normal),
		Quadruplet_indices(quadruplet_indices), Quadruplet_magnitude(quadruplet_magnitude)
	{
		// Renormalize the external field from Tesla to whatever
		for (unsigned int i = 0; i < external_field_magnitude.size(); ++i)
		{
			this->external_field_magnitude[i] = this->external_field_magnitude[i] * Constants::mu_B * mu_s[i];
		}

		this->Update_Energy_Contributions();
	}

	void Hamiltonian_Heisenberg::Update_Energy_Contributions()
	{
		this->energy_contributions_per_spin = std::vector<std::pair<std::string, scalarfield>>(0);

		// External field
		if (this->external_field_index.size() > 0)
		{
			this->energy_contributions_per_spin.push_back({"Zeeman", scalarfield(0)});
			this->idx_zeeman = this->energy_contributions_per_spin.size()-1;
		}
		else this->idx_zeeman = -1;
		// Anisotropy
		if (this->anisotropy_index.size() > 0)
		{
			this->energy_contributions_per_spin.push_back({"Anisotropy", scalarfield(0) });
			this->idx_anisotropy = this->energy_contributions_per_spin.size()-1;
		}
		else this->idx_anisotropy = -1;
		// Exchange
		if (this->Exchange_indices[0].size() > 0)
		{
			this->energy_contributions_per_spin.push_back({"Exchange", scalarfield(0) });
			this->idx_exchange = this->energy_contributions_per_spin.size()-1;
		}
		else this->idx_exchange = -1;
		// DMI
		if (this->DMI_indices[0].size() > 0)
		{
			this->energy_contributions_per_spin.push_back({"DMI", scalarfield(0) });
			this->idx_dmi = this->energy_contributions_per_spin.size()-1;
		}
		else this->idx_dmi = -1;
		// Dipole-Dipole
		if (this->DD_indices[0].size() > 0)
		{
			this->energy_contributions_per_spin.push_back({"DD", scalarfield(0) });
			this->idx_dd = this->energy_contributions_per_spin.size()-1;
		}
		else this->idx_dd = -1;
		// Quadruplet
		if (this->Quadruplet_indices[0].size() > 0)
		{
			this->energy_contributions_per_spin.push_back({"Quadruplet", scalarfield(0) });
			this->idx_quadruplet = this->energy_contributions_per_spin.size()-1;
		}
		else this->idx_quadruplet = -1;
	}


	void Hamiltonian_Heisenberg::Energy_Contributions_per_Spin(const vectorfield & spins, std::vector<std::pair<std::string, scalarfield>> & contributions)
	{
		int nos = spins.size();
		for (auto& pair : contributions)
		{
			// Allocate if not already allocated
			if (pair.second.size() != nos) pair.second = scalarfield(nos, 0);
			// Otherwise set to zero
			else for (auto& pair : contributions) Vectormath::fill(pair.second, 0);
		}
		

		// External field
		if (this->idx_zeeman >=0 ) E_Zeeman(spins, contributions[idx_zeeman].second);

		// Anisotropy
		if (this->idx_anisotropy >=0 ) E_Anisotropy(spins, contributions[idx_anisotropy].second);

		// Pairs
		//		Loop over periodicity
		for (int i_periodicity = 0; i_periodicity < 8; ++i_periodicity)
		{
			// Check if boundary conditions contain this periodicity
			if ((i_periodicity == 0)
				|| (i_periodicity == 1 && this->boundary_conditions[0])
				|| (i_periodicity == 2 && this->boundary_conditions[1])
				|| (i_periodicity == 3 && this->boundary_conditions[2])
				|| (i_periodicity == 4 && this->boundary_conditions[0] && this->boundary_conditions[1])
				|| (i_periodicity == 5 && this->boundary_conditions[0] && this->boundary_conditions[2])
				|| (i_periodicity == 6 && this->boundary_conditions[1] && this->boundary_conditions[2])
				|| (i_periodicity == 7 && this->boundary_conditions[0] && this->boundary_conditions[1] && this->boundary_conditions[2]))
			{
				//		Energies of this periodicity
				// Exchange
				if (this->idx_exchange >=0 ) E_Exchange(spins, Exchange_indices[i_periodicity], Exchange_magnitude[i_periodicity], contributions[idx_exchange].second);
				// DMI
				if (this->idx_dmi >=0 ) E_DMI(spins, DMI_indices[i_periodicity], DMI_magnitude[i_periodicity], DMI_normal[i_periodicity], contributions[idx_dmi].second);
				// DD
				if (this->idx_dd >=0 ) E_DD(spins, DD_indices[i_periodicity], DD_magnitude[i_periodicity], DD_normal[i_periodicity], contributions[idx_dd].second);
				// Quadruplet
				if (this->idx_quadruplet >=0 ) E_Quadruplet(spins, Quadruplet_indices[i_periodicity], Quadruplet_magnitude[i_periodicity], contributions[idx_quadruplet].second);
			}
		}
		
		cudaDeviceSynchronize();

		// Return
		//return this->E;
	}

	
	__global__ void CU_E_Zeeman(const Vector3 * spins, const int * external_field_index, const scalar * external_field_magnitude, const Vector3 * external_field_normal, scalar * Energy, size_t size)
	{
		for(auto idx = blockIdx.x * blockDim.x + threadIdx.x;
			idx < size;
			idx +=  blockDim.x * gridDim.x)
		{
			atomicAdd(&Energy[external_field_index[idx]], - external_field_magnitude[idx] * external_field_normal[idx].dot(spins[external_field_index[idx]]));
		}
	}
	void Hamiltonian_Heisenberg::E_Zeeman(const vectorfield & spins, scalarfield & Energy)
	{
		int size = this->external_field_index.size();
		CU_E_Zeeman<<<(size+1023)/1024, 1024>>>(spins.data(), this->external_field_index.data(), this->external_field_magnitude.data(), this->external_field_normal.data(), Energy.data(), size);
	}


	__global__ void CU_E_Anisotropy(const Vector3 * spins, const int * anisotropy_index, const scalar * anisotropy_magnitude, const Vector3 * anisotropy_normal, scalar * Energy, size_t size)
	{
		for(auto idx = blockIdx.x * blockDim.x + threadIdx.x;
			idx < size;
			idx +=  blockDim.x * gridDim.x)
		{
			atomicAdd(&Energy[anisotropy_index[idx]], - anisotropy_magnitude[idx] * std::pow(anisotropy_normal[idx].dot(spins[anisotropy_index[idx]]), 2.0));
		}
	}
	void Hamiltonian_Heisenberg::E_Anisotropy(const vectorfield & spins, scalarfield & Energy)
	{
		int size = this->anisotropy_index.size();
		CU_E_Anisotropy<<<(size+1023)/1024, 1024>>>(spins.data(), this->anisotropy_index.data(), this->anisotropy_magnitude.data(), this->anisotropy_normal.data(), Energy.data(), size);
	}


	__global__ void CU_E_Exchange(const Vector3 * spins, const indexPair * pairs, const scalar * J_ij, scalar * Energy, size_t size)
	{
		for(auto iPair = blockIdx.x * blockDim.x + threadIdx.x;
			iPair < size;
			iPair +=  blockDim.x * gridDim.x)
		{
			int ispin = pairs[iPair][0];
			int jspin = pairs[iPair][1];
			scalar sc = - 0.5 * J_ij[iPair] * spins[ispin].dot(spins[jspin]);
			atomicAdd(&Energy[ispin], sc);
			atomicAdd(&Energy[jspin], sc);
		}
	}
	void Hamiltonian_Heisenberg::E_Exchange(const vectorfield & spins, indexPairs & indices, scalarfield & J_ij, scalarfield & Energy)
	{
		int size = indices.size();
		CU_E_Exchange<<<(size+1023)/1024, 1024>>>(spins.data(), indices.data(), J_ij.data(), Energy.data(), size);
	}


	__global__ void CU_E_DMI(const Vector3 * spins, const indexPair * pairs, const scalar * DMI_magnitude, const Vector3 * DMI_normal, scalar * Energy, size_t size)
	{
		for(auto iPair = blockIdx.x * blockDim.x + threadIdx.x;
			iPair < size;
			iPair +=  blockDim.x * gridDim.x)
		{
			int ispin = pairs[iPair][0];
			int jspin = pairs[iPair][1];
			scalar sc = - 0.5 *  DMI_magnitude[iPair] * DMI_normal[iPair].dot(spins[ispin].cross(spins[jspin]));
			atomicAdd(&Energy[ispin], sc);
			atomicAdd(&Energy[jspin], sc);
		}
	}
	void Hamiltonian_Heisenberg::E_DMI(const vectorfield & spins, indexPairs & indices, scalarfield & DMI_magnitude, vectorfield & DMI_normal, scalarfield & Energy)
	{
		int size = indices.size();
		CU_E_DMI<<<(size+1023)/1024, 1024>>>(spins.data(), indices.data(), DMI_magnitude.data(), DMI_normal.data(), Energy.data(), size);
	}


	void Hamiltonian_Heisenberg::E_DD(const vectorfield & spins, indexPairs & indices, scalarfield & DD_magnitude, vectorfield & DD_normal, scalarfield & Energy)
	{
		//scalar mult = -Utility::Constants::mu_B*Utility::Constants::mu_B*1.0 / 4.0 / M_PI; // multiply with mu_B^2
		scalar mult = 0.5*0.0536814951168; // mu_0*mu_B**2/(4pi*10**-30) -- the translations are in angstr�m, so the |r|[m] becomes |r|[m]*10^-10

		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			if (DD_magnitude[i_pair] > 0.0)
			{
				Energy[indices[i_pair][0]] -= mult / std::pow(DD_magnitude[i_pair], 3.0) *
					(3 * spins[indices[i_pair][1]].dot(DD_normal[i_pair]) * spins[indices[i_pair][0]].dot(DD_normal[i_pair]) - spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]]));
				Energy[indices[i_pair][1]] -= mult / std::pow(DD_magnitude[i_pair], 3.0) *
					(3 * spins[indices[i_pair][1]].dot(DD_normal[i_pair]) * spins[indices[i_pair][0]].dot(DD_normal[i_pair]) - spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]]));
			}

		}
	}// end DipoleDipole


	void Hamiltonian_Heisenberg::E_Quadruplet(const vectorfield & spins, indexQuadruplets & indices, scalarfield & magnitude, scalarfield & Energy)
	{
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			Energy[indices[i_pair][0]] -= 0.25*magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
			Energy[indices[i_pair][1]] -= 0.25*magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
			Energy[indices[i_pair][2]] -= 0.25*magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
			Energy[indices[i_pair][3]] -= 0.25*magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
		}
	}



	void Hamiltonian_Heisenberg::Gradient(const vectorfield & spins, vectorfield & gradient)
	{
		// Set to zero
		Vectormath::fill(gradient, {0,0,0});

		// External field
		Gradient_Zeeman(gradient);

		// Anisotropy
		Gradient_Anisotropy(spins, gradient);

		// Pairs
		//		Loop over periodicity
		for (int i_periodicity = 0; i_periodicity < 8; ++i_periodicity)
		{
			// Check if boundary conditions contain this periodicity
			if ((i_periodicity == 0)
				|| (i_periodicity == 1 && this->boundary_conditions[0])
				|| (i_periodicity == 2 && this->boundary_conditions[1])
				|| (i_periodicity == 3 && this->boundary_conditions[2])
				|| (i_periodicity == 4 && this->boundary_conditions[0] && this->boundary_conditions[1])
				|| (i_periodicity == 5 && this->boundary_conditions[0] && this->boundary_conditions[2])
				|| (i_periodicity == 6 && this->boundary_conditions[1] && this->boundary_conditions[2])
				|| (i_periodicity == 7 && this->boundary_conditions[0] && this->boundary_conditions[1] && this->boundary_conditions[2]))
			{
				//		Fields of this periodicity
				// Exchange
				this->Gradient_Exchange(spins, Exchange_indices[i_periodicity], Exchange_magnitude[i_periodicity], gradient);
				// DMI
				this->Gradient_DMI(spins, DMI_indices[i_periodicity], DMI_magnitude[i_periodicity], DMI_normal[i_periodicity], gradient);
				// DD
				this->Gradient_DD(spins, DD_indices[i_periodicity], DD_magnitude[i_periodicity], DD_normal[i_periodicity], gradient);
				// Quadruplet
				this->Gradient_Quadruplet(spins, Quadruplet_indices[i_periodicity], Quadruplet_magnitude[i_periodicity], gradient);
			}
		}

		// Triplet Interactions

		// Quadruplet Interactions

		cudaDeviceSynchronize();
	}


	__global__ void CU_Gradient_Zeeman( const int * external_field_index, const scalar * external_field_magnitude, const Vector3 * external_field_normal, Vector3 * gradient, size_t size)
	{
		for(auto idx = blockIdx.x * blockDim.x + threadIdx.x;
			idx < size;
			idx +=  blockDim.x * gridDim.x)
		{
			int ispin = external_field_index[idx];
			for (int dim=0; dim<3 ; dim++)
			{
				atomicAdd(&gradient[ispin][dim], -external_field_magnitude[idx]*external_field_normal[idx][dim]);
			}
		}
	}
	void Hamiltonian_Heisenberg::Gradient_Zeeman(vectorfield & gradient)
	{
		int size = this->external_field_index.size();
		CU_Gradient_Zeeman<<<(size+1023)/1024, 1024>>>( this->external_field_index.data(), this->external_field_magnitude.data(), this->external_field_normal.data(), gradient.data(), size );
	}


	__global__ void CU_Gradient_Anisotropy(const Vector3 * spins, const int * anisotropy_index, const scalar * anisotropy_magnitude, const Vector3 * anisotropy_normal, Vector3 * gradient, size_t size)
	{
		for(auto idx = blockIdx.x * blockDim.x + threadIdx.x;
			idx < size;
			idx +=  blockDim.x * gridDim.x)
		{
			int ispin = anisotropy_index[idx];
			scalar sc = -2 * anisotropy_magnitude[idx] * anisotropy_normal[idx].dot(spins[ispin]);
			for (int dim=0; dim<3 ; dim++)
			{
				atomicAdd(&gradient[ispin][dim], sc*anisotropy_normal[idx][dim]);
			}
		}
	}
	void Hamiltonian_Heisenberg::Gradient_Anisotropy(const vectorfield & spins, vectorfield & gradient)
	{
		int size = this->anisotropy_index.size();
		CU_Gradient_Anisotropy<<<(size+1023)/1024, 1024>>>( spins.data(), this->anisotropy_index.data(), this->anisotropy_magnitude.data(), this->anisotropy_normal.data(), gradient.data(), size );
	}


	__global__ void CU_Gradient_Exchange(const Vector3 * spins, const indexPair * pairs, const scalar * J_ij, Vector3 * gradient, size_t size)
	{
		for(auto iPair = blockIdx.x * blockDim.x + threadIdx.x;
			iPair < size;
			iPair +=  blockDim.x * gridDim.x)
		{
			int ispin = pairs[iPair][0];
			int jspin = pairs[iPair][1];
			for (int dim=0; dim<3 ; dim++)
			{
				atomicAdd(&gradient[ispin][dim], -J_ij[iPair]*spins[jspin][dim]);
				atomicAdd(&gradient[jspin][dim], -J_ij[iPair]*spins[ispin][dim]);
			}
		}
	}
	void Hamiltonian_Heisenberg::Gradient_Exchange(const vectorfield & spins, indexPairs & indices, scalarfield & J_ij, vectorfield & gradient)
	{
		int size = indices.size();
		CU_Gradient_Exchange<<<(size+1023)/1024, 1024>>>( spins.data(), indices.data(), J_ij.data(), gradient.data(), size );
	}


	__global__ void CU_Gradient_DMI(const Vector3 * spins, const indexPair * pairs, const scalar * DMI_magnitude, const Vector3 * DMI_normal, Vector3 * gradient, size_t size)
	{
		for(auto iPair = blockIdx.x * blockDim.x + threadIdx.x;
			iPair < size;
			iPair +=  blockDim.x * gridDim.x)
		{
			int ispin = pairs[iPair][0];
			int jspin = pairs[iPair][1];
			Vector3 jcross = DMI_magnitude[iPair]*spins[jspin].cross(DMI_normal[iPair]);
			Vector3 icross = DMI_magnitude[iPair]*spins[ispin].cross(DMI_normal[iPair]);
			for (int dim=0; dim<3 ; dim++)
			{
				atomicAdd(&gradient[ispin][dim], -jcross[dim]);
				atomicAdd(&gradient[jspin][dim],  icross[dim]);
			}
		}
	}
	void Hamiltonian_Heisenberg::Gradient_DMI(const vectorfield & spins, indexPairs & indices, scalarfield & DMI_magnitude, vectorfield & DMI_normal, vectorfield & gradient)
	{
		int size = indices.size();
		CU_Gradient_DMI<<<(size+1023)/1024, 1024>>>( spins.data(), indices.data(), DMI_magnitude.data(), DMI_normal.data(), gradient.data(), size );
	}


	void Hamiltonian_Heisenberg::Gradient_DD(const vectorfield & spins, indexPairs & indices, scalarfield & DD_magnitude, vectorfield & DD_normal, vectorfield & gradient)
	{
		//scalar mult = Utility::Constants::mu_B*Utility::Constants::mu_B*1.0 / 4.0 / M_PI; // multiply with mu_B^2
		scalar mult = 0.0536814951168; // mu_0*mu_B**2/(4pi*10**-30) -- the translations are in angstr�m, so the |r|[m] becomes |r|[m]*10^-10
		
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			if (DD_magnitude[i_pair] > 0.0)
			{
				scalar skalar_contrib = mult / std::pow(DD_magnitude[i_pair], 3.0);
				gradient[indices[i_pair][0]] -= skalar_contrib * (3 * DD_normal[i_pair] * spins[indices[i_pair][1]].dot(DD_normal[i_pair]) - spins[indices[i_pair][1]]);
				gradient[indices[i_pair][1]] -= skalar_contrib * (3 * DD_normal[i_pair] * spins[indices[i_pair][0]].dot(DD_normal[i_pair]) - spins[indices[i_pair][0]]);
			}
		}
	}//end Field_DipoleDipole


	void Hamiltonian_Heisenberg::Gradient_Quadruplet(const vectorfield & spins, indexQuadruplets & indices, scalarfield & magnitude, vectorfield & gradient)
	{
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			gradient[indices[i_pair][0]] -= magnitude[i_pair] * spins[indices[i_pair][1]] * (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
			gradient[indices[i_pair][1]] -= magnitude[i_pair] * spins[indices[i_pair][0]] *  (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
			gradient[indices[i_pair][2]] -= magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * spins[indices[i_pair][3]];
			gradient[indices[i_pair][3]] -= magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * spins[indices[i_pair][2]];
		}
	}


	void Hamiltonian_Heisenberg::Hessian(const vectorfield & spins, MatrixX & hessian)
	{
		int nos = spins.size();

		// Set to zero
		// for (auto& h : hessian) h = 0;
		hessian.setZero();

		// Single Spin elements
		for (int alpha = 0; alpha < 3; ++alpha)
		{
			for (unsigned int i = 0; i < anisotropy_index.size(); ++i)
			{
				int idx = anisotropy_index[i];
				// scalar x = -2.0*this->anisotropy_magnitude[i] * std::pow(this->anisotropy_normal[i][alpha], 2);
				hessian(3*idx + alpha, 3*idx + alpha) += -2.0*this->anisotropy_magnitude[i]*std::pow(this->anisotropy_normal[i][alpha],2);
			}
		}

		// std::cerr << "calculated hessian" << std::endl;

		 // Spin Pair elements
		 for (int i_periodicity = 0; i_periodicity < 8; ++i_periodicity)
		 {
		 	//		Check if boundary conditions contain this periodicity
		 	if ((i_periodicity == 0)
		 		|| (i_periodicity == 1 && this->boundary_conditions[0])
		 		|| (i_periodicity == 2 && this->boundary_conditions[1])
		 		|| (i_periodicity == 3 && this->boundary_conditions[2])
		 		|| (i_periodicity == 4 && this->boundary_conditions[0] && this->boundary_conditions[1])
		 		|| (i_periodicity == 5 && this->boundary_conditions[0] && this->boundary_conditions[2])
		 		|| (i_periodicity == 6 && this->boundary_conditions[1] && this->boundary_conditions[2])
		 		|| (i_periodicity == 7 && this->boundary_conditions[0] && this->boundary_conditions[1] && this->boundary_conditions[2]))
		 	{
		 		//		Loop over pairs of this periodicity
		 		// Exchange
		 		for (unsigned int i_pair = 0; i_pair < this->Exchange_indices[i_periodicity].size(); ++i_pair)
		 		{
		 			for (int alpha = 0; alpha < 3; ++alpha)
		 			{
		 				int idx_i = 3*Exchange_indices[i_periodicity][i_pair][0] + alpha;
		 				int idx_j = 3*Exchange_indices[i_periodicity][i_pair][1] + alpha;
		 				hessian(idx_i,idx_j) += -Exchange_magnitude[i_periodicity][i_pair];
		 				hessian(idx_j,idx_i) += -Exchange_magnitude[i_periodicity][i_pair];
		 			}
		 		}
		 		// DMI
		 		for (unsigned int i_pair = 0; i_pair < this->DMI_indices[i_periodicity].size(); ++i_pair)
		 		{
		 			for (int alpha = 0; alpha < 3; ++alpha)
		 			{
		 				for (int beta = 0; beta < 3; ++beta)
		 				{
		 					int idx_i = 3*DMI_indices[i_periodicity][i_pair][0] + alpha;
		 					int idx_j = 3*DMI_indices[i_periodicity][i_pair][1] + beta;
		 					if ( (alpha == 0 && beta == 1) )
		 					{
		 						hessian(idx_i,idx_j) +=
		 							-DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][2];
		 						hessian(idx_j,idx_i) +=
		 							-DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][2];
		 					}
		 					else if ( (alpha == 1 && beta == 0) )
		 					{
		 						hessian(idx_i,idx_j) +=
		 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][2];
		 						hessian(idx_j,idx_i) +=
		 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][2];
		 					}
		 					else if ( (alpha == 0 && beta == 2) )
		 					{
		 						hessian(idx_i,idx_j) +=
		 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][1];
		 						hessian(idx_j,idx_i) +=
		 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][1];
		 					}
		 					else if ( (alpha == 2 && beta == 0) )
		 					{
		 						hessian(idx_i,idx_j) +=
		 							-DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][1];
		 						hessian(idx_j,idx_i) +=
		 							-DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][1];
		 					}
		 					else if ( (alpha == 1 && beta == 2) )
		 					{
		 						hessian(idx_i,idx_j) +=
		 							-DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][0];
		 						hessian(idx_j,idx_i) +=
		 							-DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][0];
		 					}
		 					else if ( (alpha == 2 && beta == 1) )
		 					{
		 						hessian(idx_i,idx_j) +=
		 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][0];
		 						hessian(idx_j,idx_i) +=
		 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][0];
		 					}
		 				}
		 			}
		 		}
		 //		// Dipole-Dipole
		 //		for (unsigned int i_pair = 0; i_pair < this->DD_indices[i_periodicity].size(); ++i_pair)
		 //		{
		 //			// indices
		 //			int idx_1 = DD_indices[i_periodicity][i_pair][0];
		 //			int idx_2 = DD_indices[i_periodicity][i_pair][1];
		 //			// prefactor
		 //			scalar prefactor = 0.0536814951168
		 //				* this->mu_s[idx_1] * this->mu_s[idx_2]
		 //				/ std::pow(DD_magnitude[i_periodicity][i_pair], 3);
		 //			// components
		 //			for (int alpha = 0; alpha < 3; ++alpha)
		 //			{
		 //				for (int beta = 0; beta < 3; ++beta)
		 //				{
		 //					int idx_h = idx_1 + alpha*nos + 3 * nos*(idx_2 + beta*nos);
		 //					if (alpha == beta)
		 //						hessian[idx_h] += prefactor;
		 //					hessian[idx_h] += -3.0*prefactor*DD_normal[i_periodicity][i_pair][alpha] * DD_normal[i_periodicity][i_pair][beta];
		 //				}
		 //			}
		 //		}
		 	}// end if periodicity
		 }// end for periodicity
	}

	// Hamiltonian name as string
	static const std::string name = "Heisenberg Heisenberg";
	const std::string& Hamiltonian_Heisenberg::Name() { return name; }
}

#endif